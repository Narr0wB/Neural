#include "hip/hip_runtime.h"

#include "cudalinear.h"
#include <hip/hip_runtime.h>

__device__ double square (double in)
{
    return pow(in, 2);
}

__device__ double dsigmoid (double in)
{
    return 1.0 / (1 + exp(-1 * in));
}

__device__ double dsigmoid_d (double in)
{
    return dsigmoid(in) * (1 - dsigmoid(in));
}

typedef double (*op_func) (double);

__device__ op_func funclist[NFUNCTIONS] = { dsigmoid, dsigmoid_d, square };


__global__ void mat_add(DPDOUBLE A, DPDOUBLE B, DPDOUBLE sum, size_t matRows, size_t matCols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < matRows; i += stride)
        for (size_t j = 0; j < matCols; j++)
            sum[ix(i, j, matCols)] = A[ix(i, j, matCols)] + B[ix(i, j, matCols)];
}

__global__ void mat_subtract(DPDOUBLE A, DPDOUBLE B, DPDOUBLE subtracted, size_t matRows, size_t matCols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < matRows; i += stride)
        for (size_t j = 0; j < matCols; j++)
            subtracted[ix(i, j, matCols)] = A[ix(i, j, matCols)] - B[ix(i, j, matCols)];
}

__global__ void mat_dot(DPDOUBLE A, DPDOUBLE B, DPDOUBLE dot, size_t dotRows, size_t dotCols, size_t commonDim)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < dotRows; i += stride)
        for (size_t j = 0; j < dotCols; j++)
        {
            double sum = 0;
            for (size_t k = 0; k < commonDim; k++)
                sum += A[ix(i, k, commonDim)] * B[ix(k, j, dotCols)];
            dot[ix(i, j, dotCols)] = sum;
        }
}

__global__ void mat_multiply(DPDOUBLE A, DPDOUBLE B, DPDOUBLE multiplied, size_t matRows, size_t matCols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < matRows; i += stride)
        for (size_t j = 0; j < matCols; j++)
            multiplied[ix(i, j, matCols)] = A[ix(i, j, matCols)] * B[ix(i, j, matCols)];
}

__global__ void mat_scale(DPDOUBLE A, DPDOUBLE scaled, double s, size_t matRows, size_t matCols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < matRows; i += stride)
        for (size_t j = 0; j < matCols; j++)
            scaled[ix(i, j, matCols)] = A[ix(i, j, matCols)] * s;
}

__global__ void mat_transpose(DPDOUBLE A, DPDOUBLE transposed, size_t A_rows, size_t A_cols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < A_cols; i += stride)
        for (size_t j = 0; j < A_rows; j++)
            transposed[ix(i, j, A_rows)] = A[ix(j, i, A_cols)];
}

__global__ void mat_apply(DPDOUBLE A, DPDOUBLE applied, int op_idx, size_t matRows, size_t matCols)
{
    size_t idx = threadIdx.x;
    size_t stride = blockDim.x;

    for (size_t i = idx; i < matRows; i += stride)
        for (size_t j = 0; j < matCols; j++)
            applied[ix(i, j, matCols)] = funclist[op_idx](A[ix(i, j, matCols)]);
}

// IN-DEVICE MATRIX OPERATIONS ---------------------------------------------------------------------

DMATRIX dMatAdd(DMATRIX A, DMATRIX B)
{
    if (A.rows != B.rows || 
        A.cols != B.cols)
        throw "INVALID MATRICES";

    DMATRIX sum;
    sum.rows = A.rows;
    sum.cols = A.cols;
    hipMalloc(&sum.dataPtr, (sum.rows * sum.cols) * sizeof(double));

    mat_add<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, B.dataPtr, sum.dataPtr, sum.rows, sum.cols);

    //hipDeviceSynchronize();

    return sum;
}

DMATRIX dMatSubtract(DMATRIX A, DMATRIX B)
{
    if (A.rows != B.rows || 
        A.cols != B.cols)
        throw "INVALID MATRICES";
    
    DMATRIX diff;
    diff.rows = A.rows;
    diff.cols = A.cols;
    hipMalloc(&diff.dataPtr, (diff.rows * diff.cols) * sizeof(double));

    mat_subtract<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, B.dataPtr, diff.dataPtr, diff.rows, diff.cols);

    //hipDeviceSynchronize();

    return diff;
}

DMATRIX dMatDot(DMATRIX A, DMATRIX B)
{
    if (A.cols != B.rows)
        throw "INVALID MATRICES";

    DMATRIX dot;
    dot.rows = A.rows;
    dot.cols = B.cols;
    hipMalloc(&dot.dataPtr, (dot.rows * dot.cols) * sizeof(double));

    mat_dot<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, B.dataPtr, dot.dataPtr, dot.rows, dot.cols, A.cols);

    //hipDeviceSynchronize();

    return dot;
}

DMATRIX dMatMultiply(DMATRIX A, DMATRIX B)
{
    if (A.rows != B.rows || 
        A.cols != B.cols)
        throw "INVALID MATRICES";
    
    DMATRIX mul;
    mul.rows = A.rows;
    mul.cols = A.cols;
    hipMalloc(&mul.dataPtr, (mul.rows * mul.cols) * sizeof(double));

    mat_multiply<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, B.dataPtr, mul.dataPtr, mul.rows, mul.cols);

    //hipDeviceSynchronize();

    return mul;
}

DMATRIX dMatScale(DMATRIX A, double s)
{
    if (A.rows == 0 ||
        A.cols == 0)
        throw "INVALID MATRIX";
    
    DMATRIX scal;
    scal.rows = A.rows;
    scal.cols = A.cols;
    hipMalloc(&scal.dataPtr, (scal.rows * scal.cols) * sizeof(double));

    mat_scale<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, scal.dataPtr, s, scal.rows, scal.cols);

    //hipDeviceSynchronize();

    return scal;
}

DMATRIX dMatTranspose(DMATRIX A)
{
    if (A.rows == 0 ||
        A.cols == 0)
        throw "INVALID MATRIX";
    
    DMATRIX transp;
    transp.rows = A.cols;
    transp.cols = A.rows;
    hipMalloc(&transp.dataPtr, (transp.rows * transp.cols) * sizeof(double));

    mat_transpose<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, transp.dataPtr, A.rows, A.cols);

    //hipDeviceSynchronize();

    return transp;
}

DMATRIX dMatApply(DMATRIX A, int op_idx)
{
    if (A.rows == 0 ||
        A.cols == 0)
        throw "INVALID MATRIX";

    if (op_idx > NFUNCTIONS)
        throw "INVALID FUNCTION INDEX";

    DMATRIX app;
    app.rows = A.rows;
    app.cols = A.cols;
    hipMalloc(&app.dataPtr, (app.rows * app.cols) * sizeof(double));

    mat_apply<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A.dataPtr, app.dataPtr, op_idx, app.rows, app.cols);

    //hipDeviceSynchronize();

    return app;
}

// WRAPPERS ----------------------------------------------------------------------------------------

// NOTE: The caller has to make sure that the matrices dimensions match before calling the wrapper functions, doing otherwise will lead to undefined behaviour 
void cudaw_mat_add(HPDOUBLE A_host, HPDOUBLE B_host, HPDOUBLE sum_host, size_t matRows, size_t matCols)
{
    if (A_host == nullptr ||
        B_host == nullptr ||
        sum_host == nullptr)
        return;

    DPDOUBLE A_dev;
    DPDOUBLE B_dev;
    DPDOUBLE sum_dev;

    hipMalloc(&A_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&B_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&sum_dev, (matRows * matCols)*sizeof(double));

    for (int i = 0; i < matRows; i++)
    {
        hipMemcpy(A_dev + (matCols * i), A_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_dev + (matCols * i), B_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
    }
    
    mat_add<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, B_dev, sum_dev, matRows, matCols);

    hipDeviceSynchronize();

    for (int i = 0; i < matRows; i++)
        hipMemcpy(sum_host[i], sum_dev + (matCols * i), matCols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(sum_dev);
}

void cudaw_mat_subtract(HPDOUBLE A_host, HPDOUBLE B_host, HPDOUBLE sub_host, size_t matRows, size_t matCols)
{
    if (A_host == nullptr ||
        B_host == nullptr ||
        sub_host == nullptr)
        return;

    DPDOUBLE A_dev;
    DPDOUBLE B_dev;
    DPDOUBLE sub_dev;

    hipMalloc(&A_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&B_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&sub_dev, (matRows * matCols)*sizeof(double));

    for (int i = 0; i < matRows; i++)
    {
        hipMemcpy(A_dev + (matCols * i), A_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_dev + (matCols * i), B_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
    }

    mat_subtract<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, B_dev, sub_dev, matRows, matCols);

    hipDeviceSynchronize();

    for (int i = 0; i < matRows; i++)
        hipMemcpy(sub_host[i], sub_dev + (matCols * i), matCols * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(sub_dev);
}

void cudaw_mat_dot(HPDOUBLE A_host, HPDOUBLE B_host, HPDOUBLE dot_host, size_t dot_rows, size_t dot_cols, size_t commonDim)
{
    if (A_host == nullptr ||
        B_host == nullptr ||
        dot_host == nullptr)
        return;
    
    DPDOUBLE A_dev;
    DPDOUBLE B_dev;
    DPDOUBLE dot_dev;

    hipMalloc(&A_dev, (dot_rows * commonDim)*sizeof(double));
    hipMalloc(&B_dev, (commonDim * dot_cols)*sizeof(double));
    hipMalloc(&dot_dev, (dot_rows * dot_cols)*sizeof(double));

    for (int i = 0; i < dot_rows; i++)
        hipMemcpy(A_dev + (commonDim * i), A_host[i], commonDim * sizeof(double), hipMemcpyHostToDevice);
    
    for (int i = 0; i < commonDim; i++)
        hipMemcpy(B_dev + (dot_cols * i), B_host[i], dot_cols * sizeof(double), hipMemcpyHostToDevice);

    mat_dot<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, B_dev, dot_dev, dot_rows, dot_cols, commonDim);

    hipDeviceSynchronize();

    for (int i = 0; i < dot_rows; i++)
        hipMemcpy(dot_host[i], dot_dev + (dot_cols * i), dot_cols * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(dot_dev);
}

void cudaw_mat_multiply(HPDOUBLE A_host, HPDOUBLE B_host, HPDOUBLE mul_host, size_t matRows, size_t matCols)
{
    if (A_host == nullptr ||
        B_host == nullptr ||
        mul_host == nullptr)
        return;

    DPDOUBLE A_dev;
    DPDOUBLE B_dev;
    DPDOUBLE mul_dev;

    hipMalloc(&A_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&B_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&mul_dev, (matRows * matCols)*sizeof(double));

    for (int i = 0; i < matRows; i++)
    {
        hipMemcpy(A_dev + (matCols * i), A_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(B_dev + (matCols * i), B_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);
    }
    
    mat_multiply<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, B_dev, mul_dev, matRows, matCols);

    hipDeviceSynchronize();

    for (int i = 0; i < matRows; i++)
        hipMemcpy(mul_host[i], mul_dev + (matCols * i), matCols * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(mul_dev);
}

void cudaw_mat_scale(HPDOUBLE A_host, HPDOUBLE scal_host, double s, size_t matRows, size_t matCols)
{
    if (A_host == nullptr ||
        scal_host == nullptr)
        return;

    DPDOUBLE A_dev;
    DPDOUBLE scal_dev;

    hipMalloc(&A_dev, (matRows * matCols)*sizeof(double));
    hipMalloc(&scal_dev, (matRows * matCols)*sizeof(double));

    for (int i = 0; i < matRows; i++)
        hipMemcpy(A_dev + (matCols * i), A_host[i], matCols * sizeof(double), hipMemcpyHostToDevice);

    mat_scale<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, scal_dev, s, matRows, matCols);

    hipDeviceSynchronize();

    for (int i = 0; i < matRows; i++)
        hipMemcpy(scal_host[i], scal_dev + (matCols * i), matCols * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(A_dev);
    hipFree(scal_dev);
}

void cudaw_mat_transpose(HPDOUBLE A_host, HPDOUBLE transp_host, size_t A_rows, size_t A_cols)
{
    if (A_host == nullptr ||
        transp_host == nullptr)
        return;

    DPDOUBLE A_dev;
    DPDOUBLE transp_dev;

    hipMalloc(&A_dev, (A_rows * A_cols)*sizeof(double));
    hipMalloc(&transp_dev, (A_rows * A_cols)*sizeof(double));

    for (int i = 0; i < A_rows; i++)
        hipMemcpy(A_dev + (A_cols * i), A_host[i], A_cols * sizeof(double), hipMemcpyHostToDevice);

    mat_transpose<<<KERNEL_GRID_SIZE, KERNEL_BLOCK_SIZE>>>(A_dev, transp_dev, A_rows, A_cols);

    hipDeviceSynchronize();

    for (int i = 0; i < A_cols; i++)
        hipMemcpy(transp_host[i], transp_dev + (A_rows * i), A_rows * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(A_dev);
    hipFree(transp_dev);
}

// MISC --------------------------------------------------------------------------------------------

DMATRIX copyHostToDevice(HPDOUBLE A, size_t rows, size_t cols)
{   
    DMATRIX d_mat;
    DPDOUBLE d_A;

    auto result = hipMalloc(&d_A, (rows * cols) * sizeof(double));

    if (result != hipSuccess)
        throw "Memory allocation error";

    for (int i = 0; i < rows; i++)
    {
        result = hipMemcpy(d_A + (cols * i), A[i], cols * sizeof(double), hipMemcpyHostToDevice);
        if (result != hipSuccess)
            throw "Memory copy error";
    }
    
    d_mat.dataPtr = d_A;
    d_mat.rows = rows;
    d_mat.cols = cols;

    return d_mat;
}

HPDOUBLE copyDeviceToHost(DMATRIX A)
{
    HPDOUBLE data = new double*[A.rows];
    for (int i = 0; i < A.rows; i++)
    {
        data[i] = new double[A.cols];
        auto result = hipMemcpy(data[i], A.dataPtr + (A.cols * i), A.cols * sizeof(double), hipMemcpyDeviceToHost);
        if (result != hipSuccess)
            throw "Memory copy error";
    }

    return data;
}

DMATRIX dMatCreate(size_t rows, size_t cols)
{
    DMATRIX mat;
    mat.rows = rows;
    mat.cols = cols;

    auto result = hipMalloc(&mat.dataPtr, (mat.rows * mat.cols) * sizeof(double));

    if (result != hipSuccess)
        throw "Memory allocation error";

    return mat;
}

void dMatFree(DMATRIX A)
{
    auto result = hipFree(A.dataPtr);

    if (result != hipSuccess)
        throw "Memory free error";
}